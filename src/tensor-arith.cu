#include "hip/hip_runtime.h"
#include "macros.h"
#include "tensor.h"
#include <cassert>

namespace ten {
#define DEF_MAP_KERNEL(name, op)                                               \
    KERNEL name##_kernel(ssize_t n, const float *__restrict__ in,              \
                         float *__restrict__ out) {                            \
        ssize_t i = blockIdx.x * blockDim.x + threadIdx.x;                     \
        if (i < n)                                                             \
            op;                                                                \
    }
#define DEF_MAP_OPT(name, ker_name, op)                                        \
    DEF_MAP_KERNEL(ker_name, op)                                               \
    Tensor name(const Tensor &x) {                                             \
        assert(x.device() == TensorDevice::gpu);                               \
        Tensor out(x.shape(), x.device());                                     \
        ker_name##_kernel<<<(x.size() + BLOCK_SIZE - 1) / BLOCK_SIZE,          \
                            BLOCK_SIZE>>>(x.size(), x.data(), out.data());     \
        return out;                                                            \
    }

#define DEF_BINARY_KERNEL(name, arg1, arg2, op)                                \
    KERNEL name##_kernel(ssize_t n, const float *__restrict__ arg1,            \
                         const float *__restrict__ arg2,                       \
                         float *__restrict__ out) {                            \
        ssize_t i = blockIdx.x * blockDim.x + threadIdx.x;                     \
        if (i < n)                                                             \
            op;                                                                \
    }

#define DEF_BINARY_KERNEL_EXT(name, arg1, arg2, op)                            \
    template <int xstep, int ystep>                                            \
    KERNEL name##_kernel(ssize_t n, const float *__restrict__ arg1,            \
                         const float *__restrict__ arg2,                       \
                         float *__restrict__ out) {                            \
        ssize_t i = blockIdx.x * blockDim.x + threadIdx.x;                     \
        if (i < n) {                                                           \
            ssize_t i1 = i * xstep, i2 = i * ystep;                            \
            op;                                                                \
        }                                                                      \
    }

#define DEF_BINARY_OPT(name, ker_name, arg1, arg2, op)                         \
    DEF_BINARY_KERNEL(ker_name, arg1, arg2, op)                                \
    Tensor name(const Tensor &arg1, const Tensor &arg2) {                      \
        assert(arg1.device() == TensorDevice::gpu);                            \
        assert(arg2.device() == TensorDevice::gpu);                            \
        assert(arg1.shape() == arg2.shape());                                  \
        Tensor out(arg1.shape(), arg1.device());                               \
        ker_name##_kernel<<<(arg1.size() + BLOCK_SIZE - 1) / BLOCK_SIZE,       \
                            BLOCK_SIZE>>>(arg1.size(), arg1.data(),            \
                                          arg2.data(), out.data());            \
        return out;                                                            \
    }

#define DEF_BINARY_OPT_EXT(name, ker_name, arg1, arg2, op)                     \
    DEF_BINARY_KERNEL_EXT(ker_name, arg1, arg2, op)                            \
    Tensor name(const Tensor &arg1, const Tensor &arg2) {                      \
        assert(arg1.device() == TensorDevice::gpu);                            \
        assert(arg2.device() == TensorDevice::gpu);                            \
        if (arg1.shape().empty() && arg2.shape().empty()) {                    \
            Tensor out(std::vector<ssize_t>{}, arg1.device());                 \
            ker_name##_kernel<0, 0>                                            \
                <<<(arg1.size() + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>( \
                    arg1.size(), arg1.data(), arg2.data(), out.data());        \
            return out;                                                        \
        }                                                                      \
        if (arg1.shape().empty()) {                                            \
            Tensor out(arg2.shape(), arg2.device());                           \
            ker_name##_kernel<0, 1>                                            \
                <<<(arg2.size() + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>( \
                    arg2.size(), arg1.data(), arg2.data(), out.data());        \
            return out;                                                        \
        }                                                                      \
        if (arg2.shape().empty()) {                                            \
            Tensor out(arg1.shape(), arg1.device());                           \
            ker_name##_kernel<1, 0>                                            \
                <<<(arg1.size() + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>( \
                    arg1.size(), arg1.data(), arg2.data(), out.data());        \
            return out;                                                        \
        }                                                                      \
        assert(arg1.shape() == arg2.shape());                                  \
                                                                               \
        Tensor out(arg1.shape(), arg1.device());                               \
        ker_name##_kernel<1, 1>                                                \
            <<<(arg1.size() + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(     \
                arg1.size(), arg1.data(), arg2.data(), out.data());            \
        return out;                                                            \
    }

#define DEF_FUN_OPT(f) DEF_MAP_OPT(f, f, out[i] = ::f(in[i]))

#define DEF_SCALAR_KERNEL(name, op)                                            \
    KERNEL name##_kernel(ssize_t n, const float *x, float y, float *out) {     \
        ssize_t i = blockIdx.x * blockDim.x + threadIdx.x;                     \
        if (i < n)                                                             \
            out[i] = x[i] op y;                                                \
    }

#define DEF_COMM_SCALAR_OPT(ker_name, op)                                      \
    DEF_SCALAR_KERNEL(ker_name, op)                                            \
    Tensor operator op(const Tensor &x, float y) {                             \
        assert(x.device() == TensorDevice::gpu);                               \
        Tensor out(x.shape(), x.device());                                     \
        ker_name##_kernel<<<(x.size() + BLOCK_SIZE - 1) / BLOCK_SIZE,          \
                            BLOCK_SIZE>>>(x.size(), x.data(), y, out.data());  \
        return out;                                                            \
    }                                                                          \
    Tensor operator op(float x, const Tensor &y) { return y op x; }

DEF_MAP_OPT(relu, relu, out[i] = in[i] < 0.0 ? 0.0 : in[i])
DEF_MAP_OPT(sigmoid, sigmoid, out[i] = 1.0 / (1.0 + ::exp(-in[i])))
DEF_MAP_OPT(operator-, neg, out[i] = -in[i])
DEF_BINARY_OPT(relu_grad, relu_grad, x, grad,
               out[i] = x[i] < 0.0 ? 0.0 : grad[i])
DEF_BINARY_OPT(sigmoid_grad, sigmoid_grad, y, grad,
               out[i] = grad[i] * y[i] * (1.0 - y[i]))
DEF_FUN_OPT(exp)
DEF_FUN_OPT(log)
DEF_FUN_OPT(log2)
DEF_FUN_OPT(sqrt)
DEF_MAP_OPT(square, square, out[i] = in[i] * in[i])
DEF_BINARY_OPT_EXT(operator+, pointwise_plus, x, y, out[i] = x[i1] + y[i2])
DEF_BINARY_OPT_EXT(operator-, pointwise_minus, x, y, out[i] = x[i1] - y[i2])
DEF_BINARY_OPT_EXT(operator*, pointwise_mult, x, y, out[i] = x[i1] * y[i2])
DEF_BINARY_OPT_EXT(operator/, pointwise_div, x, y, out[i] = x[i1] / y[i2])
DEF_COMM_SCALAR_OPT(scalar_add, +)
DEF_COMM_SCALAR_OPT(scalar_mult, *)
} // namespace ten
